#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : gpu_euclidean_clustering.cu
 Author      : AnhNV91
 Version     : 1.0
 Description : Clustering analysis using Euclidean distance and single linkage
 ============================================================================
 */

#include "gpu_euclidean_clustering.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/fill.h>

#include <time.h>
#include <sys/time.h>

#define MAX_SHARED_SIZE 2048
#define BLOCK_SIZE_X 1024

//#define SERIAL 1

inline void gassert(hipError_t err_code, const char *file, int line)
{
  if (err_code != hipSuccess)
  {
    fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
    hipDeviceReset();
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

GpuEuclideanCluster::GpuEuclideanCluster()
{
  x_ = NULL;
  y_ = NULL;
  z_ = NULL;

  size_ = 0;
  threshold_ = 0;
  cluster_indices_ = NULL;
  cluster_indices_host_ = NULL;
  min_cluster_pts_ = 0;
  max_cluster_pts_ = 1000000000;
  cluster_num_ = 0;
}

void GpuEuclideanCluster::setInputPoints(float *x, float *y, float *z, int size)
{
  size_ = size;
  checkCudaErrors(hipMalloc(&x_, size_ * sizeof(float)));
  checkCudaErrors(hipMalloc(&y_, size_ * sizeof(float)));
  checkCudaErrors(hipMalloc(&z_, size_ * sizeof(float)));

  checkCudaErrors(hipMemcpy(x_, x, size_ * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(y_, y, size_ * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(z_, z, size_ * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&cluster_indices_, size_ * sizeof(int)));
  cluster_indices_host_ = (int *) malloc(size_ * sizeof(int));
}

void GpuEuclideanCluster::setThreshold(double threshold)
{
  threshold_ = threshold;
}

void GpuEuclideanCluster::setMinClusterPts(int min_cluster_pts)
{
  min_cluster_pts_ = min_cluster_pts;
}

void GpuEuclideanCluster::setMaxClusterPts(int max_cluster_pts)
{
  max_cluster_pts_ = max_cluster_pts;
}

/* Initially, each point is assigned to an individual cluster.
 *
 */

extern "C" __global__ void pclEuclideanInitialize(int *cluster_indices, int size)
{
  for (int index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x)
    cluster_indices[index] = index;
}

/* Connected component labeling points at GPU block thread level.
 * Input list of points is divided into multiple smaller groups.
 * Each group of point is assigned to a block of GPU thread.
 * Each thread in a block handles one point in the group. It iterates over
 * points in the group and compare the distance between the current point A
 * and the point B it has to handle.
 *
 * If the distance between A and B is less than the threshold, then those
 * two points belong to a same connected component and the cluster_changed
 * is marked by 1.
 *
 * A synchronization is called to make sure all thread in the block finish A
 * before moving to the update phase.
 * After finishing checking cluster_changed, threads update the cluster
 * index of all points. If a thread has cluster_changed is 1, then the corresponding
 * cluster of the point it is handling is changed to the cluster of B. Otherwise
 * the original cluster of A remains unchanged.
 *
 * Another synchronization is called before all threads in the block move to
 * other points after done checking A.
 *
 * After this kernel finishes, all points in each block are labeled.
 */
extern "C" __global__ void blockLabelling(float *x, float *y, float *z, int *cluster_indices, int size, float threshold)
{
  int block_start = blockIdx.x * blockDim.x;
  int block_end = (block_start + blockDim.x <= size) ? (block_start + blockDim.x) : size;
  int row = threadIdx.x + block_start;
  __shared__ int local_offset[BLOCK_SIZE_X];
  __shared__ float local_x[BLOCK_SIZE_X];
  __shared__ float local_y[BLOCK_SIZE_X];
  __shared__ float local_z[BLOCK_SIZE_X];
  __shared__ int local_cluster_changed[BLOCK_SIZE_X];

  if (row < block_end)
  {
    local_offset[threadIdx.x] = threadIdx.x;
    local_x[threadIdx.x] = x[row];
    local_y[threadIdx.x] = y[row];
    local_z[threadIdx.x] = z[row];
    __syncthreads();

    for (int column = block_start; column < block_end; column++)
    {
      float tmp_x = local_x[threadIdx.x] - local_x[column - block_start];
      float tmp_y = local_y[threadIdx.x] - local_y[column - block_start];
      float tmp_z = local_z[threadIdx.x] - local_z[column - block_start];
      int column_offset = local_offset[column - block_start];
      int row_offset = local_offset[threadIdx.x];

      local_cluster_changed[threadIdx.x] = 0;
      __syncthreads();

      if (row > column && column_offset != row_offset && norm3df(tmp_x, tmp_y, tmp_z) < threshold)
        local_cluster_changed[row_offset] = 1;
      __syncthreads();

      local_offset[threadIdx.x] = (local_cluster_changed[row_offset] == 1) ? column_offset : row_offset;
      __syncthreads();
    }

    __syncthreads();

    int new_cluster = cluster_indices[block_start + local_offset[threadIdx.x]];

    __syncthreads();

    cluster_indices[row] = new_cluster;
  }
}

/* These kernels are used to collect remained clusters after each labeling phase.
 *
 * Basically, in each labeling phases, several clusters are merged together.
 *
 * The first kernel scans over the cluster_indices array and marks the cluster_mark
 * element corresponding with the cluster of the current point by 1. If a cluster
 * does not exists in the current phase (which means it was merged to some other
 * clusters), then its cluster_mark is 0.
 *
 * The second kernel scans over the original cluster_indices again and copy those
 * indices to new location on the target_clusters.
 */
extern "C" __global__ void clusterMark(int *cluster_list, int *cluster_mark, int size)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x)
    cluster_mark[cluster_list[i]] = 1;
}

extern "C" __global__ void
clusterCollector(int *old_cluster_list, int *new_cluster_list, int *cluster_location, int size)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x)
    new_cluster_list[cluster_location[old_cluster_list[i]]] = old_cluster_list[i];
}

/* Create a cluster matrix.
 *
 * A cluster matrix is to record the relationship between each pair
 * of clusters. If a pair of cluster x and y are connected, then
 * the matrix element [x][y] are 1. Otherwise it is 0. Notice that
 * only the lower half of the matrix is used.
 *
 * To build this matrix, each GPU thread handles one point A, iterates
 * over all points B, and compare distance between A and B. Assuming
 * that A belongs to a cluster x, and B belongs to cluster y. If their
 * distance is less than the threshold, then the matrix element [x][y]
 * is set to 1.
 */
extern "C" __global__ void
buildClusterMatrix(float *x, float *y, float *z, int *cluster_indices, int *cluster_matrix, int *cluster_offset,
                   int size, int cluster_num, float threshold)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  __shared__ float local_x[BLOCK_SIZE_X];
  __shared__ float local_y[BLOCK_SIZE_X];
  __shared__ float local_z[BLOCK_SIZE_X];

  if (index>size)
    return;

  for (int column = index; column < size; column += stride)
  {
    local_x[threadIdx.x] = x[column];
    local_y[threadIdx.x] = y[column];
    local_z[threadIdx.x] = z[column];
    int column_cluster = cluster_indices[column];
    int cc_offset = cluster_offset[column_cluster];

    __syncthreads();

    for (int row = 0; row < column; row++)
    {
      float tmp_x = x[row] - local_x[threadIdx.x];
      float tmp_y = y[row] - local_y[threadIdx.x];
      float tmp_z = z[row] - local_z[threadIdx.x];
      int row_cluster = cluster_indices[row];
      int rc_offset = cluster_offset[row_cluster];

      if (row_cluster != column_cluster && norm3df(tmp_x, tmp_y, tmp_z) < threshold)
        cluster_matrix[rc_offset * cluster_num + cc_offset] = 1;
    }
    __syncthreads();
  }
}

/* Merge clusters based on the cluster_matrix.
 *
 * This merge process is done per block. The input list of clusters
 * are divided into smaller chunks to be handled by GPU blocks.
 *
 * Each thread in a block handles one row of the matrix and iterates
 * over all columns of the matrix. A synchronization per each iteration
 * is needed to make sure all threads done merging clusters in the
 * current column before moving to the next column.
 *
 * In each iteration, each thread check if the cluster corresponding
 * with the current row is connected to the cluster corresponding to the
 * current column. If so, then the cluster of the row is changed (merged)
 * to the cluster of the column.
 */
extern "C" __global__ void mergeClusters(int *cluster_matrix, int *cluster_list, int cluster_num)
{
  int row_start = blockIdx.x * blockDim.x;
  int row_end = (row_start + blockDim.x <= cluster_num) ? row_start + blockDim.x : cluster_num;
  int col = row_start + threadIdx.x;
  __shared__ int local_changed[BLOCK_SIZE_X];
  __shared__ int local_offset[BLOCK_SIZE_X];

  /* The cluster matrix is symmetric, so the
   * number of rows and columns are the same
   */
  if (col < row_end)
  {
    local_offset[threadIdx.x] = threadIdx.x;

    __syncthreads();

    for (int row = row_start; row < row_end; row++)
    {
      int col_offset = local_offset[threadIdx.x];
      int row_offset = local_offset[row - row_start];

      local_changed[threadIdx.x] = 0;
      __syncthreads();

      if (row < col && row_offset != col_offset && (cluster_matrix[row * cluster_num + col] == 1))
        local_changed[col_offset] = 1;
      __syncthreads();

      local_offset[threadIdx.x] = (local_changed[col_offset] == 1) ? row_offset : col_offset;
      __syncthreads();
    }

    __syncthreads();

    int new_cluster = cluster_list[row_start + local_offset[threadIdx.x]];

    __syncthreads();
    cluster_list[col] = new_cluster;
  }
}

/* Reflex the change in the cluster merging step
 * to cluster indices of all input points.
 *
 * Clusters of input points are changed to the target clusters
 * corresponding with their source clusters.
 */
extern "C" __global__ void reflexClusterChanges(int *cluster_indices, int *cluster_offset, int *cluster_list, int size)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x)
    cluster_indices[i] = cluster_list[cluster_offset[cluster_indices[i]]];
}

/* Rebuild cluster matrix after merging clusters.
 *
 * After several cluster are merged together, the number of clusters
 * reduces and the cluster matrix needs to be rebuilt.
 *
 * Each thread iterate over rows of one column of the source matrix.
 * If a element [x][y] of the source matrix is 1, then the element
 * [m][n] of the target matrix, in which m and n are the
 * new clusters of x and y, is set to 1.
 */
extern "C" __global__ void
rebuildClusterMatrix(int *old_cluster_matrix, int *new_clusters, int *new_cluster_matrix, int *new_cluster_offset,
                     int old_size, int new_size)
{
  for (int column = threadIdx.x + blockIdx.x * blockDim.x; column < old_size; column += blockDim.x * gridDim.x)
  {
    for (int row = 0; row < column; row++)
    {
      int new_row = new_cluster_offset[new_clusters[row]];
      int new_column = new_cluster_offset[new_clusters[column]];

      if (old_cluster_matrix[row * old_size + column] == 1)
        new_cluster_matrix[new_row * new_size + new_column] = 1;
    }
  }
}

/* Perform exclusive scan on the input array using
 * thurst's scan.
 *
 * The variable 'sum' records the last element of
 * the array after being scanned.
 */
void GpuEuclideanCluster::exclusiveScan(int *input, int ele_num, int *sum)
{
  thrust::device_ptr<int> dev_ptr(input);

  thrust::exclusive_scan(dev_ptr, dev_ptr + ele_num, dev_ptr);
  checkCudaErrors(hipDeviceSynchronize());

  *sum = *(dev_ptr + ele_num - 1);
}

/* Reset the cluster indexes in the point cloud from 0.
 *
 * After merging, the cluster indexes of points are still large. Collecting
 * those large indexes is either time-consuming (without using hash) or
 * wasting memory space (using hash). By reset the cluster indexes from 0,
 * we can use hashing to collect those indexes with the space complexity equal
 * to the number of clusters.
 */
extern "C" __global__ void resetClusterIndexes(int *cluster_indices, int *cluster_offset, int size)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += blockDim.x * gridDim.x)
  {
    int old_cluster = cluster_indices[i];

    cluster_indices[i] = cluster_offset[old_cluster];
  }
}


/* Calculate the cluster indices of input points.
 *
 * Initially, the cluster index of the point at index ith
 * is set to i. This method merges cluster indices
 * of points that belong to same clusters.
 *
 * Result of this method is stored at cluster_indices_host_.
 */
void GpuEuclideanCluster::extractClustersOld()
{
  int block_x, grid_x;

  block_x = (size_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : size_;
  grid_x = (size_ - 1) / block_x + 1;

  int *cluster_offset;
  int cluster_num, old_cluster_num;

  pclEuclideanInitialize << < grid_x, block_x >> > (cluster_indices_, size_);
  checkCudaErrors(hipDeviceSynchronize());

  old_cluster_num = cluster_num = size_;

  checkCudaErrors(hipMalloc(&cluster_offset, (size_ + 1) * sizeof(int)));
  checkCudaErrors(hipMemset(cluster_offset, 0, (size_ + 1) * sizeof(int)));
  blockLabelling << < grid_x, block_x >> > (x_, y_, z_, cluster_indices_, size_, threshold_);
  clusterMark << < grid_x, block_x >> > (cluster_indices_, cluster_offset, size_);
  exclusiveScan(cluster_offset, size_ + 1, &cluster_num);

  int *cluster_list, *new_cluster_list, *tmp;

  checkCudaErrors(hipMalloc(&cluster_list, cluster_num * sizeof(int)));
  clusterCollector << < grid_x, block_x >> > (cluster_indices_, cluster_list, cluster_offset, size_);
  checkCudaErrors(hipDeviceSynchronize());

  int *cluster_matrix;
  int *new_cluster_matrix;

  checkCudaErrors(hipMalloc(&cluster_matrix, cluster_num * cluster_num * sizeof(int)));
  checkCudaErrors(hipMemset(cluster_matrix, 0, cluster_num * cluster_num * sizeof(int)));
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMalloc(&new_cluster_list, cluster_num * sizeof(int)));

  buildClusterMatrix << < grid_x, block_x >> >
                                  (x_, y_, z_, cluster_indices_, cluster_matrix, cluster_offset, size_, cluster_num, threshold_);
  checkCudaErrors(hipDeviceSynchronize());

  int block_x2 = 0, grid_x2 = 0;


  /* Loop until there is no change in the number of clusters */
  do
  {
    old_cluster_num = cluster_num;
    block_x2 = (cluster_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : cluster_num;
    grid_x2 = (cluster_num - 1) / block_x2 + 1;

    mergeClusters << < grid_x2, block_x2 >> > (cluster_matrix, cluster_list, cluster_num);
    reflexClusterChanges << < grid_x, block_x >> > (cluster_indices_, cluster_offset, cluster_list, size_);
    checkCudaErrors(hipMemset(cluster_offset, 0, (size_ + 1) * sizeof(int)));
    clusterMark << < grid_x2, block_x2 >> > (cluster_list, cluster_offset, cluster_num);
    exclusiveScan(cluster_offset, size_ + 1, &cluster_num);

    if (grid_x2 == 1 && cluster_num == old_cluster_num)
      break;

    clusterCollector << < grid_x2, block_x2 >> > (cluster_list, new_cluster_list, cluster_offset, old_cluster_num);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMalloc(&new_cluster_matrix, cluster_num * cluster_num * sizeof(int)));
    checkCudaErrors(hipMemset(new_cluster_matrix, 0, cluster_num * cluster_num * sizeof(int)));
    rebuildClusterMatrix << < grid_x2, block_x2 >> >
                                       (cluster_matrix, cluster_list, new_cluster_matrix, cluster_offset, old_cluster_num, cluster_num);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(cluster_matrix));
    cluster_matrix = new_cluster_matrix;
    tmp = cluster_list;
    cluster_list = new_cluster_list;
    new_cluster_list = tmp;
  } while (1);

  cluster_num_ = cluster_num;

  resetClusterIndexes << < grid_x, block_x >> > (cluster_indices_, cluster_offset, size_);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(cluster_indices_host_, cluster_indices_, size_ * sizeof(int), hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(cluster_matrix));
  checkCudaErrors(hipFree(cluster_list));
  checkCudaErrors(hipFree(new_cluster_list));
  checkCudaErrors(hipFree(cluster_offset));
}

extern "C" __global__ void mergeSelfClusters(int *cluster_matrix, int *cluster_list, int cluster_num, bool *changed)
{
  int row_start = blockIdx.x * blockDim.x;
  int row_end = (row_start + blockDim.x <= cluster_num) ? row_start + blockDim.x : cluster_num;
  int col = row_start + threadIdx.x;
  __shared__ int local_changed[BLOCK_SIZE_X];
  __shared__ int local_offset[BLOCK_SIZE_X];
  bool block_changed = false;

  if (col < row_end)
  {
    local_offset[threadIdx.x] = threadIdx.x;

    __syncthreads();

    for (int row = row_start; row < row_end; row++)
    {
      int col_offset = local_offset[threadIdx.x];
      int row_offset = local_offset[row - row_start];

      local_changed[threadIdx.x] = 0;
      __syncthreads();

      if (row < col && row_offset != col_offset && (cluster_matrix[row * cluster_num + col] == 1))
      {
        local_changed[col_offset] = 1;
        block_changed = true;
      }
      __syncthreads();

      local_offset[threadIdx.x] = (local_changed[col_offset] == 1) ? row_offset : col_offset;
      __syncthreads();
    }

    __syncthreads();

    int new_cluster = cluster_list[row_start + local_offset[threadIdx.x]];

    __syncthreads();

    cluster_list[col] = new_cluster;


    __syncthreads();
    if (block_changed)
      *changed = true;
  }
}

/* Merge clusters from different blocks of points.
 *
 * The relationship of those clusters are expressed by a cluster matrix.
 * The merge is done by assigning each thread in a block of GPU threads
 * to move from top to bottom of the matrix and check if there are any
 * 1 element in the matrix.
 *
 * This kernel only merge matrices that staying in a same diagonal of a
 * group of matrix. The index of the diagonal is indicated by shift_level.
 */

extern "C" __global__ void mergeInterClusters(int *cluster_matrix, int *cluster_list,
                                              int shift_level,
                                              int base_row, int base_column,
                                              int sub_matrix_row, int sub_matrix_col,
                                              int sub_matrix_offset_row, int sub_matrix_offset_col,
                                              int cluster_num, bool *changed)
{
  int col_start = (base_column + (blockIdx.x / sub_matrix_col) * sub_matrix_offset_col +
                   (blockIdx.x + shift_level - sub_matrix_col * ((blockIdx.x + shift_level) / sub_matrix_col))) *
                  blockDim.x;
  int col_end = (col_start + blockDim.x <= cluster_num) ? col_start + blockDim.x : cluster_num;
  int row_start = (base_row + (blockIdx.x / sub_matrix_row) * sub_matrix_offset_row +
                   (blockIdx.x - sub_matrix_row * (blockIdx.x / sub_matrix_row))) * blockDim.x;
  int row_end = (row_start + blockDim.x <= cluster_num) ? row_start + blockDim.x : cluster_num;
  int col = col_start + threadIdx.x;

  __shared__ int local_changed[BLOCK_SIZE_X];
  __shared__ int local_offset[BLOCK_SIZE_X];
  bool block_changed = false;

  if (col < col_end)
  {
    local_offset[threadIdx.x] = threadIdx.x;
    __syncthreads();

    for (int row = row_start; row < row_end; row++)
    {
      int col_offset = local_offset[threadIdx.x];
      int row_offset = local_offset[row - row_start];

      local_changed[threadIdx.x] = 0;
      __syncthreads();

      if (row_offset != col_offset && cluster_matrix[row * cluster_num + col] == 1)
      {
        local_changed[col_offset] = 1;
        block_changed = true;
      }
      __syncthreads();

      local_offset[threadIdx.x] = (local_changed[col_offset] == 1) ? row_offset : col_offset;
      __syncthreads();
    }

    __syncthreads();
    int new_cluster = cluster_list[col_start + local_offset[threadIdx.x]];

    __syncthreads();
    cluster_list[col] = new_cluster;

    if (block_changed)
      *changed = true;
  }
}

/* Checking if two individual blocks have any clusters that intersect.
 *
 * If there are, then the diagonal index that the block belongs to is
 * recorded in changed_diag. All blocks in the same diagonal are merged
 * in the next step.
 */
extern "C" __global__ void clustersIntersecCheck(int *cluster_matrix, int *changed_diag,
                                                 int base_row, int base_column,
                                                 int sub_matrix_row, int sub_matrix_col,
                                                 int sub_matrix_offset_row, int sub_matrix_offset_col,
                                                 int cluster_num)
{
  //Thinking about using % or not
  int col_idx = (blockIdx.x / sub_matrix_col) * sub_matrix_offset_col + (blockIdx.x % sub_matrix_col);
  int row_idx = (blockIdx.x / sub_matrix_row) * sub_matrix_offset_row + (blockIdx.y % sub_matrix_col);

  int col_start = (base_column + col_idx) * blockDim.x;
  int col_end = (col_start + blockDim.x <= cluster_num) ? col_start + blockDim.x : cluster_num;
  int row_start = (base_row + row_idx) * blockDim.x;
  int row_end = (row_start + blockDim.x <= cluster_num) ? row_start + blockDim.x : cluster_num;
  int col = col_start + threadIdx.x;
  int diag_offset = (col_idx > row_idx) ? col_idx - row_idx : col_idx + row_idx;

  if (col < col_end && col_start <= col_end && row_start <= row_end)
  {
    for (int row = row_start; row < row_end; row++)
    {
      if (cluster_matrix[row * cluster_num + col] == 1)
      {
        *changed_diag = diag_offset;
        break;
      }
    }
  }
}

/* Extract clusters of points.
 *
 * This method can handle the case with sparse points (distance between points
 * are larger than threshold), which may lead to infinite loop in the first method.
 */

void GpuEuclideanCluster::extractClusters()
{
  int block_x, grid_x;

  block_x = (size_ > BLOCK_SIZE_X) ? BLOCK_SIZE_X : size_;
  grid_x = (size_ - 1) / block_x + 1;

  int *cluster_offset;
  int cluster_num, old_cluster_num;

  pclEuclideanInitialize << < grid_x, block_x >> > (cluster_indices_, size_);
  checkCudaErrors(hipDeviceSynchronize());

  old_cluster_num = cluster_num = size_;

  checkCudaErrors(hipMalloc(&cluster_offset, (size_ + 1) * sizeof(int)));
  checkCudaErrors(hipMemset(cluster_offset, 0, (size_ + 1) * sizeof(int)));
  blockLabelling << < grid_x, block_x >> > (x_, y_, z_, cluster_indices_, size_, threshold_);
  clusterMark << < grid_x, block_x >> > (cluster_indices_, cluster_offset, size_);
  exclusiveScan(cluster_offset, size_ + 1, &cluster_num);

  int *cluster_list, *new_cluster_list, *tmp;

  checkCudaErrors(hipMalloc(&cluster_list, cluster_num * sizeof(int)));
  clusterCollector << < grid_x, block_x >> > (cluster_indices_, cluster_list, cluster_offset, size_);
  checkCudaErrors(hipDeviceSynchronize());

  int *cluster_matrix;
  int *new_cluster_matrix;

  checkCudaErrors(hipMalloc(&cluster_matrix, cluster_num * cluster_num * sizeof(int)));
  checkCudaErrors(hipMemset(cluster_matrix, 0, cluster_num * cluster_num * sizeof(int)));
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMalloc(&new_cluster_list, cluster_num * sizeof(int)));

  buildClusterMatrix << < grid_x, block_x >> >
                                  (x_, y_, z_, cluster_indices_, cluster_matrix, cluster_offset, size_, cluster_num, threshold_);
  checkCudaErrors(hipDeviceSynchronize());

  int block_x2 = 0, grid_x2 = 0;

  bool *changed;

  checkCudaErrors(hipHostMalloc(&changed, sizeof(bool)));

#ifndef SERIAL
  int *changed_diag;

  checkCudaErrors(hipHostMalloc(&changed_diag, sizeof(int)));
#endif

  int max_base_row = 0;

  do
  {
    *changed = false;
    block_x2 = (cluster_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : cluster_num;
    grid_x2 = (cluster_num - 1) / block_x2 + 1;

    mergeSelfClusters << < grid_x2, block_x2 >> > (cluster_matrix, cluster_list, cluster_num, changed);
    checkCudaErrors(hipDeviceSynchronize());

    int base_row = 1, base_column = 0;
    int sub_matrix_offset_row = 2, sub_matrix_offset_col = 2;
    int sub_matrix_row = 1, sub_matrix_col = 1;
    int sub_matrix_num;
    int max_rows = grid_x2;

    max_base_row = base_row;

    while (!(*changed) && cluster_num > BLOCK_SIZE_X && base_row * BLOCK_SIZE_X < cluster_num &&
           base_column < cluster_num)
    {

      sub_matrix_num = (cluster_num - base_row - 1) / sub_matrix_offset_row + 1;
      block_x2 = BLOCK_SIZE_X;
      grid_x2 = sub_matrix_num * sub_matrix_col;

#ifdef SERIAL
      //Merge clusters in each sub-matrix by moving from top to bottom of the similarity sub-matrix
      for (int shift_level = 0; !(*changed) && shift_level < sub_matrix_col; shift_level++) {
        mergeInterClusters<<<grid_x2, block_x2>>>(cluster_matrix, cluster_list,
                                shift_level,
                                base_row, base_column,
                                sub_matrix_row, sub_matrix_col,
                                sub_matrix_offset_row, sub_matrix_offset_col,
                                cluster_num, changed);
        checkCudaErrors(hipDeviceSynchronize());
      }
#else
      int grid_y2 = sub_matrix_row;

      dim3 block_size(block_x2, 1, 1);
      dim3 grid_size(grid_x2, grid_y2, 1);

      *changed_diag = -1;

      clustersIntersecCheck << < grid_size, block_size >> > (cluster_matrix, changed_diag,
        base_row, base_column,
        sub_matrix_row, sub_matrix_col,
        sub_matrix_offset_row, sub_matrix_offset_col,
        cluster_num);
      checkCudaErrors(hipDeviceSynchronize());

      if (*changed_diag > 0)
      {
        //Merge clusters in sub-matrix that stay in the changed_diag diagonal by moving from top to bottom of the matrix.
        mergeInterClusters << < grid_x2, block_x2 >> > (cluster_matrix, cluster_list, *changed_diag,
          base_row, base_column,
          sub_matrix_row, sub_matrix_col,
          sub_matrix_offset_row, sub_matrix_offset_col,
          cluster_num, changed);
        checkCudaErrors(hipDeviceSynchronize());
      }

#endif
      base_row += sub_matrix_row;
      sub_matrix_row = (sub_matrix_row * 2 + base_row < max_rows) ? sub_matrix_row * 2 : max_rows - base_row;
      sub_matrix_col *= 2;
      sub_matrix_offset_row *= 2;
      sub_matrix_offset_col *= 2;
    }

    max_base_row = base_row;

    if (*changed)
    {
      reflexClusterChanges << < grid_x, block_x >> > (cluster_indices_, cluster_offset, cluster_list, size_);
      checkCudaErrors(hipMemset(cluster_offset, 0, (size_ + 1) * sizeof(int)));

      block_x2 = (cluster_num > BLOCK_SIZE_X) ? BLOCK_SIZE_X : cluster_num;
      grid_x2 = (cluster_num - 1) / block_x2 + 1;

      clusterMark << < grid_x2, block_x2 >> > (cluster_list, cluster_offset, cluster_num);

      old_cluster_num = cluster_num;
      exclusiveScan(cluster_offset, size_ + 1, &cluster_num);
      clusterCollector << < grid_x2, block_x2 >> > (cluster_list, new_cluster_list, cluster_offset, old_cluster_num);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipMalloc(&new_cluster_matrix, cluster_num * cluster_num * sizeof(int)));
      checkCudaErrors(hipMemset(new_cluster_matrix, 0, cluster_num * cluster_num * sizeof(int)));
      rebuildClusterMatrix << < grid_x2, block_x2 >> >
                                         (cluster_matrix, cluster_list, new_cluster_matrix, cluster_offset, old_cluster_num, cluster_num);
      checkCudaErrors(hipDeviceSynchronize());

      checkCudaErrors(hipFree(cluster_matrix));
      cluster_matrix = new_cluster_matrix;
      tmp = cluster_list;
      cluster_list = new_cluster_list;
      new_cluster_list = tmp;
    }
  } while (*changed && max_base_row < cluster_num);

  cluster_num_ = cluster_num;

  //Reset all cluster indexes to make them start from 0
  resetClusterIndexes << < grid_x, block_x >> > (cluster_indices_, cluster_offset, size_);
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(cluster_indices_host_, cluster_indices_, size_ * sizeof(int), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(cluster_matrix));
  checkCudaErrors(hipFree(cluster_list));
  checkCudaErrors(hipFree(new_cluster_list));
  checkCudaErrors(hipFree(cluster_offset));
  checkCudaErrors(hipHostFree(changed));
#ifndef SERIAL
  checkCudaErrors(hipHostFree(changed_diag));
#endif
}

/* Collect points that belong to same clusters and put them together.
 *
 * The output is a vector whose each element contains indexes of points
 * that belong to a same clusters.
 */
std::vector <GpuEuclideanCluster::GClusterIndex> GpuEuclideanCluster::getOutput()
{
  std::vector <GClusterIndex> cluster_indices(cluster_num_);

  for (unsigned int i = 0; i < cluster_indices.size(); i++)
    cluster_indices[i].index_value = -1;

  for (int i = 0; i < size_; i++)
  {
    cluster_indices[cluster_indices_host_[i]].points_in_cluster.push_back(i);
    cluster_indices[cluster_indices_host_[i]].index_value = cluster_indices_host_[i];
  }

  for (unsigned int i = 0; i < cluster_indices.size();)
  {
    int number_of_pts = cluster_indices[i].points_in_cluster.size();

    if (number_of_pts < min_cluster_pts_ || number_of_pts > max_cluster_pts_)
      cluster_indices.erase(cluster_indices.begin() + i);
    else
      i++;
  }

  return cluster_indices;
}

/* Generate sparse points.
 * The number of points is fixed at 10000.
 * Cannot afford more (e.g. 100 000 points) since
 * GPU memory is not enough for a matrix with 10 billions cells.
 */
GpuEuclideanCluster::SamplePointListXYZ GpuEuclideanCluster::generateSample()
{
  GpuEuclideanCluster::SamplePointListXYZ output;

  output.size = 10000;

  output.x = (float *) malloc(sizeof(float) * output.size);
  output.y = (float *) malloc(sizeof(float) * output.size);
  output.z = (float *) malloc(sizeof(float) * output.size);

  output.x[0] = 0;
  output.y[0] = 0;
  output.z[0] = 0;

  for (int i = 1; i < output.size; i++)
  {
    output.x[i] = (i % 3 == 0) ? output.x[i - 1] + threshold_ + 1 : output.x[i - 1];
    output.y[i] = (i % 3 == 1) ? output.y[i - 1] + threshold_ + 1 : output.y[i - 1];
    output.z[i] = (i % 3 == 2) ? output.z[i - 1] + threshold_ + 1 : output.z[i - 1];
  }

  return output;
}

GpuEuclideanCluster::~GpuEuclideanCluster()
{
  checkCudaErrors(hipFree(x_));
  checkCudaErrors(hipFree(y_));
  checkCudaErrors(hipFree(z_));
  checkCudaErrors(hipFree(cluster_indices_));
  free(cluster_indices_host_);
}

